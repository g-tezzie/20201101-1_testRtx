
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>

// shmem
static const int buflen = 100e6;
extern char ioRaw[buflen];
extern short int raw[2][buflen / 4];
extern short int rawo[2][buflen / 4];


__global__ void addKernel(short int* c, const short int* a, const short int* b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(short int* c, const short int* a, const short int* b, unsigned int size)
{
    short int* dev_a = 0;
    short int* dev_b = 0;
    short int* dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(dev_c[0]));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(dev_a[0]));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(dev_b[0]));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(dev_a[0]), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(dev_b[0]), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel <<< 1, size >>> (dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(c[0]), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}



int maine()
{
    const int blocksize = 1000;
	for (int i = 0; i < 1000; i++) {
        int ptr = i * blocksize;
		// Add vectors in parallel.
		hipError_t cudaStatus = addWithCuda(&rawo[0][ptr], &raw[0][ptr], &raw[1][ptr], blocksize);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addWithCuda failed!");
			return 1;
		}
	}
	// NOTE: C-K C-F for "Format Selection"

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }


    return 0;
}

